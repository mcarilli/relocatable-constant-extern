#include "hip/hip_runtime.h"
#include "client1.cuh"

template<typename FD>
__global__ void client1_kernel(int* a) {
  *a = FD::get_np0();
}

template<typename FD> void client1(int* a) {
  client1_kernel<FD><<<1, 1>>>(a);  
}

template void client1<fd>(int* a);
